#include "hip/hip_runtime.h"
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "postprocess.h"


__device__ float sigmoid_gpu(const float x) { return 1.0f / (1.0f + expf(-x)); }

__global__ void BEVDecodeObjectKernel(const int map_size,         // 40000
                                   const float score_thresh,   // 0.1
                                //    const int nms_pre_max_size, // 4096
                                   const float x_start,
                                   const float y_start,
                                   const float x_step,
                                   const float y_step,
                                   const int output_h,
                                   const int output_w,
                                   const int downsample_size,
                                   const int num_class_in_task,
                                   const int cls_range,
                                   const float* reg,
                                   const float* hei,
                                   const float* dim,
                                   const float* rot,
                                   const float* vel,
                                   const float* cls,
                                   float* res_box,
                                   float* res_conf,
                                   int* res_cls,
                                   int* res_box_num,
                                   float* rescale_factor){  // 根据置信度，初筛，筛选后有res_box_num个box，不超过nms_pre_max_size 4096
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= map_size) return;

    float max_score = cls[idx]; // 初始化为task的第一个类
    int label = cls_range;      // 初始化为task的第一个类
    for (int i = 1; i < num_class_in_task; ++i) {
        float cur_score = cls[idx + i * map_size];
        if (cur_score > max_score){
            max_score = cur_score;
            label = i + cls_range;
        }
    }

    int coor_x = idx % output_h;  //
    int coor_y = idx / output_w;  //

    float conf = sigmoid_gpu(max_score); // 计算置信度
    if (conf > score_thresh){
        int cur_valid_box_id = atomicAdd(res_box_num, 1);
        res_box[cur_valid_box_id * kBoxBlockSize + 0] = 
            (reg[idx + 0 * map_size] + coor_x) * x_step + x_start;
        res_box[cur_valid_box_id * kBoxBlockSize + 1] = 
            (reg[idx + 1 * map_size] + coor_y) * y_step + y_start;
        res_box[cur_valid_box_id * kBoxBlockSize + 2] = hei[idx];
        res_box[cur_valid_box_id * kBoxBlockSize + 3] = 
                                expf(dim[idx + 0 * map_size]) * rescale_factor[label]; // nms scale
        res_box[cur_valid_box_id * kBoxBlockSize + 4] = 
                                expf(dim[idx + 1 * map_size]) * rescale_factor[label];
        res_box[cur_valid_box_id * kBoxBlockSize + 5] = 
                                expf(dim[idx + 2 * map_size]) * rescale_factor[label];
        res_box[cur_valid_box_id * kBoxBlockSize + 6] = atan2f(rot[idx], rot[idx + map_size]);
        res_box[cur_valid_box_id * kBoxBlockSize + 7] = vel[idx];
        res_box[cur_valid_box_id * kBoxBlockSize + 8] = vel[idx + map_size];


        res_conf[cur_valid_box_id] = conf;
        res_cls[cur_valid_box_id] = label;
    }
}

PostprocessGPU::PostprocessGPU(const int _class_num, 
                               const float _score_thresh,
                               const float _nms_thresh, 
                               const int _nms_pre_maxnum,
                               const int _nms_post_maxnum, 
                               const int _down_sample, 
                               const int _output_h, 
                               const int _output_w, 
                               const float _x_step, 
                               const float _y_step,
                               const float _x_start, 
                               const float _y_start,
                               const std::vector<int>& _class_num_pre_task,
                               const std::vector<float>& _nms_rescale_factor) :
                               class_num(_class_num),
                               score_thresh(_score_thresh),
                               nms_thresh(_nms_thresh), 
                               nms_pre_maxnum(_nms_pre_maxnum),
                               nms_post_maxnum(_nms_post_maxnum), 
                               down_sample(_down_sample),
                               output_h(_output_h), 
                               output_w(_output_w), 
                               x_step(_x_step),
                               y_step(_y_step), 
                               x_start(_x_start), 
                               y_start(_y_start),
                               map_size(output_h * output_w),
                               class_num_pre_task(_class_num_pre_task),
                               nms_rescale_factor(_nms_rescale_factor),
                               task_num(_class_num_pre_task.size()){

    CHECK_CUDA(hipMalloc((void**)&boxes_dev, kBoxBlockSize * map_size * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&score_dev, map_size * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&cls_dev, map_size * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&sorted_indices_dev, map_size * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&valid_box_num, sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&nms_rescale_factor_dev, class_num * sizeof(float)));

    CHECK_CUDA(hipHostMalloc((void**)&boxes_host, kBoxBlockSize * map_size * sizeof(float)));
    CHECK_CUDA(hipHostMalloc((void**)&score_host, nms_pre_maxnum * sizeof(float)));
    CHECK_CUDA(hipHostMalloc((void**)&cls_host, map_size * sizeof(float)));
    CHECK_CUDA(hipHostMalloc((void**)&sorted_indices_host, nms_pre_maxnum * sizeof(int)));
    CHECK_CUDA(hipHostMalloc((void**)&keep_data_host, nms_pre_maxnum * sizeof(long)));

    CHECK_CUDA(hipMemcpy(nms_rescale_factor_dev, nms_rescale_factor.data(),
                                        class_num * sizeof(float), hipMemcpyHostToDevice));

    iou3d_nms.reset(new Iou3dNmsCuda(output_h, output_w, nms_thresh));


    for(auto i = 0; i < nms_rescale_factor.size(); i++){
        printf("%.2f%c", nms_rescale_factor[i], i == nms_rescale_factor.size() - 1 ? '\n' : ' ');
    }

}
PostprocessGPU::~PostprocessGPU(){
    CHECK_CUDA(hipFree(boxes_dev));
    CHECK_CUDA(hipFree(score_dev));
    CHECK_CUDA(hipFree(cls_dev));
    CHECK_CUDA(hipFree(sorted_indices_dev));
    CHECK_CUDA(hipFree(valid_box_num));
    CHECK_CUDA(hipFree(nms_rescale_factor_dev));

    CHECK_CUDA(hipHostFree(boxes_host));
    CHECK_CUDA(hipHostFree(score_host));
    CHECK_CUDA(hipHostFree(cls_host));
    CHECK_CUDA(hipHostFree(sorted_indices_host));
    CHECK_CUDA(hipHostFree(keep_data_host));
}




void PostprocessGPU::DoPostprocess(void ** const bev_buffer, std::vector<Box>& out_detections){

    // bev_buffer : BEV_feat, reg_0, hei_0, dim_0, rot_0, vel_0, heatmap_0, reg_1 ...
    const int task_num = class_num_pre_task.size();
    int cur_start_label = 0;
    for(int i = 0; i < task_num; i++){
        float* reg = (float*)bev_buffer[i * 6 + 1];     // 2 x 128 x 128
        float* hei = (float*)bev_buffer[i * 6 + 2];     // 1 x 128 x 128
        float* dim = (float*)bev_buffer[i * 6 + 3];     // 3 x 128 x 128
        float* rot = (float*)bev_buffer[i * 6 + 4];     // 2 x 128 x 128
        float* vel = (float*)bev_buffer[i * 6 + 5];     // 2 x 128 x 128
        float* heatmap = (float*)bev_buffer[i * 6 + 6]; // c x 128 x 128

        dim3 grid(DIVUP(map_size, NUM_THREADS));
        CHECK_CUDA(hipMemset(valid_box_num, 0, sizeof(int)));
        BEVDecodeObjectKernel<<<grid, NUM_THREADS>>>(map_size, score_thresh, 
                                         x_start, y_start, x_step, y_step, output_h,
                                         output_w, down_sample, class_num_pre_task[i],
                                         cur_start_label, reg, hei, dim, rot, 
                                         vel, 
                                         heatmap,
                                         boxes_dev, score_dev, cls_dev, valid_box_num,
                                         nms_rescale_factor_dev);

        /*
        此时 boxes_dev, score_dev, cls_dev 有 valid_box_num 个元素，可能大于nms_pre_maxnum, 而且是无序排列的
        */ 
        int box_num_pre = 0;
        CHECK_CUDA(hipMemcpy(&box_num_pre, valid_box_num, sizeof(int), hipMemcpyDeviceToHost));

        thrust::sequence(thrust::device, sorted_indices_dev, sorted_indices_dev + box_num_pre);
        thrust::sort_by_key(thrust::device, score_dev, score_dev + box_num_pre, 
                            sorted_indices_dev, thrust::greater<float>());
        // 此时 score_dev 是降序排列的，而 sorted_indices_dev 索引着原顺序，
        // 即 sorted_indices_dev[i] = j; i:现在的位置，j:原索引;  j:[0, map_size)


        box_num_pre = std::min(box_num_pre, nms_pre_maxnum);

        int box_num_post = iou3d_nms->DoIou3dNms(box_num_pre, boxes_dev, 
                                                        sorted_indices_dev, keep_data_host);

        box_num_post = std::min(box_num_post, nms_post_maxnum);


        CHECK_CUDA(hipMemcpy(sorted_indices_host, sorted_indices_dev, box_num_pre * sizeof(int),
                                                                    hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(boxes_host, boxes_dev, kBoxBlockSize * map_size * sizeof(float),
                                                                    hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(score_host, score_dev, box_num_pre * sizeof(float), 
                                                                    hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(cls_host, cls_dev, map_size * sizeof(float), 
                                                                    hipMemcpyDeviceToHost));


        for (auto j = 0; j < box_num_post; j++) {
            int k = keep_data_host[j];
            int idx = sorted_indices_host[k];
            Box box;
            box.x = boxes_host[idx * kBoxBlockSize + 0];
            box.y = boxes_host[idx * kBoxBlockSize + 1];
            box.z = boxes_host[idx * kBoxBlockSize + 2];
            box.l = boxes_host[idx * kBoxBlockSize + 3] / nms_rescale_factor[cls_host[idx]];
            box.w = boxes_host[idx * kBoxBlockSize + 4] / nms_rescale_factor[cls_host[idx]];
            box.h = boxes_host[idx * kBoxBlockSize + 5] / nms_rescale_factor[cls_host[idx]];
            box.r = boxes_host[idx * kBoxBlockSize + 6];
            box.vx = boxes_host[idx * kBoxBlockSize + 7];
            box.vy = boxes_host[idx * kBoxBlockSize + 8];

            box.label = cls_host[idx];
            box.score = score_host[k];
            box.z -= box.h * 0.5; // bottom height
            out_detections.push_back(box);
        }
        
        cur_start_label += class_num_pre_task[i];
    }
}
