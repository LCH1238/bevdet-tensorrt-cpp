#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gatherbev_plugin.h"
#include "common.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>


namespace nvinfer1
{
// class GatherBEVPlugin
GatherBEVPlugin::GatherBEVPlugin(const std::string &name):
    name_(name){
}

GatherBEVPlugin::GatherBEVPlugin(const std::string &name, const void *buffer, size_t length):
    name_(name){
    memcpy(&m_, buffer, sizeof(m_));
}

GatherBEVPlugin::~GatherBEVPlugin(){
}

IPluginV2DynamicExt *GatherBEVPlugin::clone() const noexcept {
    auto p = new GatherBEVPlugin(name_, &m_, sizeof(m_));
    p->setPluginNamespace(namespace_.c_str());
    return p;
}

int32_t GatherBEVPlugin::getNbOutputs() const noexcept {
    return 1;
}
 
DataType GatherBEVPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, 
                                                                int32_t nbInputs) const noexcept {
    return inputTypes[0];  // 与adj_feat一致
}

DimsExprs GatherBEVPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, 
                                        int32_t nbInputs, IExprBuilder &exprBuilder) noexcept {
  // input[0] == adj_feat   8*80 x 128 x 128
  // input[1] == curr_feat  80 * 128 * 128
  // input[2] == flag       1

    DimsExprs ret;
    ret.nbDims = inputs[0].nbDims + 1;

    ret.d[0] = exprBuilder.constant(1);
    ret.d[1] = exprBuilder.operation(DimensionOperation::kSUM, *inputs[0].d[0], *inputs[1].d[0]);
    ret.d[2] = inputs[0].d[1];
    ret.d[3] = inputs[0].d[2];

    return ret; 
}

bool GatherBEVPlugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut,
                                                    int32_t nbInputs, int32_t nbOutputs) noexcept {
    // adj_feat    curr_feat   out
    if(pos == 0 || pos == 1 || pos == 3){
        return (inOut[pos].type == DataType::kFLOAT || inOut[pos].type == DataType::kHALF) &&
                inOut[pos].format == TensorFormat::kLINEAR;
    }
    else if(pos == 2){
        return inOut[pos].type == DataType::kINT32 && inOut[pos].format == TensorFormat::kLINEAR;
    }
    return false;
}

void GatherBEVPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, 
                                    const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept {
    return;
}

size_t GatherBEVPlugin::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, 
                                const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept {
    return 0;
}

int32_t GatherBEVPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc,
    const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept {
    // input[0] == adj_feat   8*80 x 128 x 128
    // input[1] == curr_feat  80 * 128 * 128
    // input[2] == flag       1

    int flag = 0;
    CHECK_CUDA(hipMemcpy(&flag, inputs[2], sizeof(int), hipMemcpyDeviceToHost));

    int feat_step = inputDesc[1].dims.d[0] * inputDesc[1].dims.d[1] * inputDesc[1].dims.d[2];
    int adj_num = inputDesc[0].dims.d[0] / inputDesc[1].dims.d[0];

    switch (int(outputDesc[0].type))
    {
    case int(DataType::kFLOAT):
        if(!flag){
            for(int i = 0; i < adj_num + 1; i++){
                CHECK_CUDA(hipMemcpy((float*)outputs[0] + i * feat_step, inputs[1], 
                                        feat_step * sizeof(float), hipMemcpyDeviceToDevice));
            }
            // printf("flag %d\n", flag);
        }
        else{
            CHECK_CUDA(hipMemcpy(outputs[0], inputs[1], feat_step * sizeof(float), hipMemcpyDeviceToDevice));
            CHECK_CUDA(hipMemcpy((float*)outputs[0] + feat_step, inputs[0], adj_num * feat_step * sizeof(float),
                                    hipMemcpyDeviceToDevice));
            // printf("flag %d\n", flag);
        }
        break;
    case int(DataType::kHALF):
        if(!flag){
            for(int i = 0; i < adj_num + 1; i++){
                CHECK_CUDA(hipMemcpy((__half*)outputs[0] + i * feat_step, inputs[1], 
                                        feat_step * sizeof(__half), hipMemcpyDeviceToDevice));
            }
        }
        else{
            CHECK_CUDA(hipMemcpy(outputs[0], inputs[1], feat_step * sizeof(__half), hipMemcpyDeviceToDevice));
            CHECK_CUDA(hipMemcpy((__half*)outputs[0] + feat_step, inputs[0], adj_num * feat_step * sizeof(__half),
                                    hipMemcpyDeviceToDevice));
        }
        break;
    default: // should NOT be here
        printf("\tUnsupport datatype!\n");
    }
    return 0;
}

void GatherBEVPlugin::destroy() noexcept {
    delete this;
    return;
}

int32_t GatherBEVPlugin::initialize() noexcept {
    return 0;
}

void GatherBEVPlugin::terminate() noexcept {
    return;
}

size_t GatherBEVPlugin::getSerializationSize() const noexcept {
    return sizeof(m_);
}

void GatherBEVPlugin::serialize(void *buffer) const noexcept {
    memcpy(buffer, &m_, sizeof(m_));
    return;
}

void GatherBEVPlugin::setPluginNamespace(const char *pluginNamespace) noexcept {
    namespace_ = pluginNamespace;
    return;
}

const char *GatherBEVPlugin::getPluginNamespace() const noexcept {
    return namespace_.c_str();
}

const char *GatherBEVPlugin::getPluginType() const noexcept {
    return PLUGIN_NAME;
}

const char *GatherBEVPlugin::getPluginVersion() const noexcept {
    return PLUGIN_VERSION;
}

void GatherBEVPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, 
                                                        IGpuAllocator *gpuAllocator) noexcept {
    return;
}

void GatherBEVPlugin::detachFromContext() noexcept {
    return;
}

// class GatherBEVPluginCreator
PluginFieldCollection    GatherBEVPluginCreator::fc_ {};
std::vector<PluginField> GatherBEVPluginCreator::attr_;

GatherBEVPluginCreator::GatherBEVPluginCreator() {
    attr_.clear();
    fc_.nbFields = attr_.size();
    fc_.fields   = attr_.data();
}

GatherBEVPluginCreator::~GatherBEVPluginCreator() {
}


IPluginV2DynamicExt *GatherBEVPluginCreator::createPlugin(const char *name, 
                                    const PluginFieldCollection *fc) noexcept {
    GatherBEVPlugin *pObj = new GatherBEVPlugin(name);
    pObj->setPluginNamespace(namespace_.c_str());
    return pObj;
}

IPluginV2DynamicExt *GatherBEVPluginCreator::deserializePlugin(const char *name, 
                                        const void *serialData, size_t serialLength) noexcept {
    GatherBEVPlugin *pObj = new GatherBEVPlugin(name, serialData, serialLength);
    pObj->setPluginNamespace(namespace_.c_str());
    return pObj;
}

void GatherBEVPluginCreator::setPluginNamespace(const char *pluginNamespace) noexcept {
    namespace_ = pluginNamespace;
    return;
}

const char *GatherBEVPluginCreator::getPluginNamespace() const noexcept {
    return namespace_.c_str();
}

const char *GatherBEVPluginCreator::getPluginName() const noexcept {
    return PLUGIN_NAME;
}

const char *GatherBEVPluginCreator::getPluginVersion() const noexcept {
    return PLUGIN_VERSION;
}

const PluginFieldCollection *GatherBEVPluginCreator::getFieldNames() noexcept {
    return &fc_;
}

REGISTER_TENSORRT_PLUGIN(GatherBEVPluginCreator);

} // namespace nvinfer1
