#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "bevpool_plugin.h"
#include "common.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

// kernel for GPU
template<typename T>
__global__ void bev_pool_v2_kernel(int c, int n_intervals, int map_size,
                                  const T *__restrict__ depth,
                                  const T *__restrict__ feat,
                                  const int *__restrict__ ranks_depth,
                                  const int *__restrict__ ranks_feat,
                                  const int *__restrict__ ranks_bev,
                                  const int *__restrict__ interval_starts,
                                  const int *__restrict__ interval_lengths,
                                  T * __restrict__ out) {
    // 进入到一个kernel的都是一个bevgrid要计算的特征的某一维度
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = idx / c;    // bev grid index
    int cur_c = idx % c;    // channel index
    if (index >= n_intervals) return;
    int interval_start = interval_starts[index];  
    int interval_length = interval_lengths[index];  
    T psum = 0;
    const T * cur_depth;
    const T * cur_feat;
    for(int i = 0; i < interval_length; i++){
        cur_depth = depth + ranks_depth[interval_start+i];            // 指向 深度概率值
        cur_feat = feat + ranks_feat[interval_start+i] * c + cur_c;   // 指向 cur_c通道图像特征值
        psum += *cur_feat * *cur_depth;
    }

    const int* cur_rank = ranks_bev + interval_start;  // 指向 某bevgrid
    // float* cur_out = out + *cur_rank * c + cur_c;   // b x h x w x c
    T* cur_out = out + cur_c * map_size + *cur_rank;      // b x c x h x w

    *cur_out = psum;
}

namespace nvinfer1
{
// class BEVPoolPlugin
BEVPoolPlugin::BEVPoolPlugin(const std::string &name, int bev_h, int bev_w):
    name_(name){
    m_.bev_h = bev_h;
    m_.bev_w = bev_w;
}

BEVPoolPlugin::BEVPoolPlugin(const std::string &name, const void *buffer, size_t length):
    name_(name){
    memcpy(&m_, buffer, sizeof(m_));
}

BEVPoolPlugin::~BEVPoolPlugin(){
}

IPluginV2DynamicExt *BEVPoolPlugin::clone() const noexcept {
    auto p = new BEVPoolPlugin(name_, &m_, sizeof(m_));
    p->setPluginNamespace(namespace_.c_str());
    return p;
}

int32_t BEVPoolPlugin::getNbOutputs() const noexcept {
    return 1;
}
 
DataType BEVPoolPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, 
                                                                int32_t nbInputs) const noexcept {
    return inputTypes[0];  // 与mean一致
}

DimsExprs BEVPoolPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, 
                                        int32_t nbInputs, IExprBuilder &exprBuilder) noexcept {
  // input[0] == depth->kFLOAT
  // input[1] == feat->kFLOAT
  // input[2] == ranks_depth->kINT32
  // input[3] == ranks_feat->kINT32
  // input[4] == ranks_bev->kINT32
  // input[5] == interval_starts->kINT32
  // input[6] == interval_lengths->kINT32

    DimsExprs ret;
    ret.nbDims = inputs[1].nbDims - 1;
    ret.d[0] = inputs[1].d[3];
    ret.d[1] = exprBuilder.constant(m_.bev_h);
    ret.d[2] = exprBuilder.constant(m_.bev_w);
    
    return ret;  // FIXME
}

bool BEVPoolPlugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut,
                                                    int32_t nbInputs, int32_t nbOutputs) noexcept {
    // depth       feat        out
    if(pos == 0 || pos == 1 || pos == 7){
        return (inOut[pos].type == DataType::kFLOAT || inOut[pos].type == DataType::kHALF) &&
                inOut[pos].format == TensorFormat::kLINEAR;
    }
    else{
        return inOut[pos].type == DataType::kINT32 && inOut[pos].format == TensorFormat::kLINEAR;
    }
    return false;
}

void BEVPoolPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, 
                                    const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept {
    return;
}

size_t BEVPoolPlugin::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, 
                                const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept {
    return 0;
}

int32_t BEVPoolPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc,
    const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept {
    // input[0] == depth->kFLOAT
    // input[1] == feat->kFLOAT
    // input[2] == ranks_depth->kINT32
    // input[3] == ranks_feat->kINT32
    // input[4] == ranks_bev->kINT32
    // input[5] == interval_starts->kINT32
    // input[6] == interval_lengths->kINT32

    int channel = inputDesc[1].dims.d[3];
    int n_intervals = inputDesc[5].dims.d[0];
    int map_size = m_.bev_h * m_.bev_w;

    // dim3 grid((int)ceil((float)(n_intervals * channel / NUM_THREADS)));
    dim3 grid(CEIL_DIVIDE(n_intervals * channel, NUM_THREADS));
    dim3 block(NUM_THREADS);

    switch (int(outputDesc[0].type))
    {
    case int(DataType::kFLOAT):
        bev_pool_v2_kernel<<<grid, block, 0, stream>>>(
                                                    channel, 
                                                    n_intervals,
                                                    map_size,
                                                    reinterpret_cast<const float *>(inputs[0]),
                                                    reinterpret_cast<const float *>(inputs[1]),
                                                    reinterpret_cast<const int *>(inputs[2]),
                                                    reinterpret_cast<const int *>(inputs[3]),
                                                    reinterpret_cast<const int *>(inputs[4]),
                                                    reinterpret_cast<const int *>(inputs[5]),
                                                    reinterpret_cast<const int *>(inputs[6]),
                                                    reinterpret_cast<float *>(outputs[0]));
        break;
    case int(DataType::kHALF):
        bev_pool_v2_kernel<<<grid, block, 0, stream>>>(
                                                    channel, 
                                                    n_intervals,
                                                    map_size,
                                                    reinterpret_cast<const __half *>(inputs[0]),
                                                    reinterpret_cast<const __half *>(inputs[1]),
                                                    reinterpret_cast<const int *>(inputs[2]),
                                                    reinterpret_cast<const int *>(inputs[3]),
                                                    reinterpret_cast<const int *>(inputs[4]),
                                                    reinterpret_cast<const int *>(inputs[5]),
                                                    reinterpret_cast<const int *>(inputs[6]),
                                                    reinterpret_cast<__half *>(outputs[0]));
        break;
    default: // should NOT be here
        printf("\tUnsupport datatype!\n");
    }
    return 0;
}

void BEVPoolPlugin::destroy() noexcept {
    delete this;
    return;
}

int32_t BEVPoolPlugin::initialize() noexcept {
    return 0;
}

void BEVPoolPlugin::terminate() noexcept {
    return;
}

size_t BEVPoolPlugin::getSerializationSize() const noexcept {
    return sizeof(m_);
}

void BEVPoolPlugin::serialize(void *buffer) const noexcept {
    memcpy(buffer, &m_, sizeof(m_));
    return;
}

void BEVPoolPlugin::setPluginNamespace(const char *pluginNamespace) noexcept {
    namespace_ = pluginNamespace;
    return;
}

const char *BEVPoolPlugin::getPluginNamespace() const noexcept {
    return namespace_.c_str();
}

const char *BEVPoolPlugin::getPluginType() const noexcept {
    return PLUGIN_NAME;
}

const char *BEVPoolPlugin::getPluginVersion() const noexcept {
    return PLUGIN_VERSION;
}

void BEVPoolPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, 
                                                        IGpuAllocator *gpuAllocator) noexcept {
    return;
}

void BEVPoolPlugin::detachFromContext() noexcept {
    return;
}

// class BEVPoolPluginCreator
PluginFieldCollection    BEVPoolPluginCreator::fc_ {};
std::vector<PluginField> BEVPoolPluginCreator::attr_;

BEVPoolPluginCreator::BEVPoolPluginCreator() {
    attr_.clear();
    attr_.emplace_back(PluginField("bev_h", nullptr, PluginFieldType::kINT32, 1));
    attr_.emplace_back(PluginField("bev_w", nullptr, PluginFieldType::kINT32, 1));

    fc_.nbFields = attr_.size();
    fc_.fields   = attr_.data();
}

BEVPoolPluginCreator::~BEVPoolPluginCreator() {
}


IPluginV2DynamicExt *BEVPoolPluginCreator::createPlugin(const char *name, 
                                    const PluginFieldCollection *fc) noexcept {
    const PluginField *fields = fc->fields;

    int bev_h = -1;
    int bev_w = -1;

    for (int i = 0; i < fc->nbFields; ++i){
        if(std::string(fc->fields[i].name) == std::string("bev_h")){
            bev_h = *reinterpret_cast<const int *>(fc->fields[i].data);
        }
        else if(std::string(fc->fields[i].name) == std::string("bev_w")){
            bev_w = *reinterpret_cast<const int *>(fc->fields[i].data);
        }
    }
    BEVPoolPlugin *pObj = new BEVPoolPlugin(name, bev_h, bev_w);
    pObj->setPluginNamespace(namespace_.c_str());
    return pObj;
}

IPluginV2DynamicExt *BEVPoolPluginCreator::deserializePlugin(const char *name, 
                                        const void *serialData, size_t serialLength) noexcept {
    BEVPoolPlugin *pObj = new BEVPoolPlugin(name, serialData, serialLength);
    pObj->setPluginNamespace(namespace_.c_str());
    return pObj;
}

void BEVPoolPluginCreator::setPluginNamespace(const char *pluginNamespace) noexcept {
    namespace_ = pluginNamespace;
    return;
}

const char *BEVPoolPluginCreator::getPluginNamespace() const noexcept {
    return namespace_.c_str();
}

const char *BEVPoolPluginCreator::getPluginName() const noexcept {
    return PLUGIN_NAME;
}

const char *BEVPoolPluginCreator::getPluginVersion() const noexcept {
    return PLUGIN_VERSION;
}

const PluginFieldCollection *BEVPoolPluginCreator::getFieldNames() noexcept {
    return &fc_;
}

REGISTER_TENSORRT_PLUGIN(BEVPoolPluginCreator);

} // namespace nvinfer1
