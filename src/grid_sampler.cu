#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved
// modified from
// https://github.com/pytorch/pytorch/blob/ec683299ebabf297a3504c76248d37be830e4342/aten/src/ATen/native/cuda/GridSampler.cuh
// and
// https://github.com/pytorch/pytorch/blob/ec683299ebabf297a3504c76248d37be830e4342/aten/src/ATen/native/cuda/GridSampler.cu


#include "grid_sampler.cuh"


// Unnormalizes a coordinate from the -1 to +1 scale to its pixel index value,
// where we view each pixel as an area between (idx - 0.5) and (idx + 0.5).
// if align_corners: -1 and +1 get sent to the centers of the corner pixels
//     -1 --> 0
//     +1 --> (size - 1)
//     scale_factor = (size - 1) / 2
// if not align_corners: -1 and +1 get sent to the image edges
//     -1 --> -0.5
//     +1 --> (size - 1) + 0.5 == size - 0.5
//     scale_factor = size / 2
template <typename scalar_t>
static __forceinline__ __device__ scalar_t grid_sampler_unnormalize(scalar_t coord, int size,
                                                                    bool align_corners) {
  if (align_corners) {
    // unnormalize coord from [-1, 1] to [0, size - 1]
    return ((coord + 1.f) / 2) * (size - 1);
  } else {
    // unnormalize coord from [-1, 1] to [-0.5, size - 0.5]
    return ((coord + 1.f) * size - 1) / 2;
  }
}

// Clips coordinates to between 0 and clip_limit - 1
template <typename scalar_t>
static __forceinline__ __device__ scalar_t clip_coordinates(scalar_t in, int clip_limit) {
  return ::min(static_cast<scalar_t>(clip_limit - 1), ::max(in, static_cast<scalar_t>(0)));
}

// Reflects coordinates until they fall between low and high (inclusive).
// The bounds are passed as twice their value so that half-integer values
// can be represented as ints.
template <typename scalar_t>
static __forceinline__ __device__ scalar_t reflect_coordinates(scalar_t in, int twice_low,
                                                               int twice_high) {
  if (twice_low == twice_high) {
    return static_cast<scalar_t>(0);
  }
  scalar_t min = static_cast<scalar_t>(twice_low) / 2;
  scalar_t span = static_cast<scalar_t>(twice_high - twice_low) / 2;
  in = ::fabs(in - min);
  // `fmod` returns same sign as `in`, which is positive after the `fabs` above.
  scalar_t extra = ::fmod(in, span);
  int flips = static_cast<int>(::floor(in / span));
  if (flips % 2 == 0) {
    return extra + min;
  } else {
    return span - extra + min;
  }
}

template <typename scalar_t>
static __forceinline__ __device__ scalar_t safe_downgrade_to_int_range(scalar_t x) {
  // -100.0 does not have special meaning. This is just to make sure
  // it's not within_bounds_2d or within_bounds_3d, and does not cause
  // undefined behavior. See #35506.
  if (x > INT_MAX - 1 || x < INT_MIN || !::isfinite(static_cast<double>(x)))
    return static_cast<scalar_t>(-100.0);
  return x;
}

// Computes the pixel source index value for a grid coordinate
template <typename scalar_t>
static __forceinline__ __device__ scalar_t grid_sampler_compute_source_index(
    scalar_t coord, int size, GridSamplerPadding padding_mode, bool align_corners) {
  coord = grid_sampler_unnormalize(coord, size, align_corners);
  if (padding_mode == GridSamplerPadding::Border) {
    // clip coordinates to image borders
    coord = clip_coordinates(coord, size);
  } else if (padding_mode == GridSamplerPadding::Reflection) {
    // reflect coordinates by image borders
    if (align_corners) {
      coord = reflect_coordinates(coord, 0, 2 * (size - 1));
    } else {
      coord = reflect_coordinates(coord, -1, 2 * size - 1);
    }
    // clip coordinates to image borders
    coord = clip_coordinates(coord, size);
  }

  coord = safe_downgrade_to_int_range(coord);
  return coord;
}

static __forceinline__ __device__ bool within_bounds_2d(int h, int w, int H, int W) {
  return h >= 0 && h < H && w >= 0 && w < W;
}


template <typename scalar_t>
__global__ void grid_sampler_2d_kernel(const int nthreads, const scalar_t *input,
                                       const scalar_t *grid, scalar_t *output,
                                       TensorDesc input_desc, TensorDesc grid_desc,
                                       TensorDesc output_desc,
                                       const GridSamplerPadding padding_mode, bool align_corners) {
  int C = input_desc.shape[1];
  int inp_H = input_desc.shape[2];
  int inp_W = input_desc.shape[3];
  int out_H = grid_desc.shape[1];
  int out_W = grid_desc.shape[2];
  int inp_sN = input_desc.stride[0];
  int inp_sC = input_desc.stride[1];
  int inp_sH = input_desc.stride[2];
  int inp_sW = input_desc.stride[3];
  int grid_sN = grid_desc.stride[0];
  int grid_sH = grid_desc.stride[1];
  int grid_sW = grid_desc.stride[2];
  int grid_sCoor = grid_desc.stride[3];
  int out_sN = output_desc.stride[0];
  int out_sC = output_desc.stride[1];
  int out_sH = output_desc.stride[2];
  int out_sW = output_desc.stride[3];

  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_W;
    const int h = (index / out_W) % out_H;
    const int n = index / (out_H * out_W);
    const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

    // get the corresponding input x, y coordinates from grid
    scalar_t ix = grid[grid_offset];
    scalar_t iy = grid[grid_offset + grid_sCoor];

    ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
    iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);


    // get NE, NW, SE, SW pixel values from (x, y)
    int ix_nw = static_cast<int>(::floor(ix));
    int iy_nw = static_cast<int>(::floor(iy));
    int ix_ne = ix_nw + 1;
    int iy_ne = iy_nw;
    int ix_sw = ix_nw;
    int iy_sw = iy_nw + 1;
    int ix_se = ix_nw + 1;
    int iy_se = iy_nw + 1;

    // get surfaces to each neighbor:
    scalar_t nw = (ix_se - ix) * (iy_se - iy);
    scalar_t ne = (ix - ix_sw) * (iy_sw - iy);
    scalar_t sw = (ix_ne - ix) * (iy - iy_ne);
    scalar_t se = (ix - ix_nw) * (iy - iy_nw);

    // calculate bilinear weighted pixel value and set output pixel
    auto inp_ptr_NC = input + n * inp_sN;
    auto out_ptr_NCHW = output + n * out_sN + h * out_sH + w * out_sW;
    for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
      *out_ptr_NCHW = static_cast<scalar_t>(0);
      if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
        *out_ptr_NCHW += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
      }
      if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
        *out_ptr_NCHW += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
      }
      if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
        *out_ptr_NCHW += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
      }
      if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
        *out_ptr_NCHW += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
      }
    }
    
  }
}


void create_desc(const int *dims, int nb_dims, TensorDesc &desc) {
  memcpy(&desc.shape[0], dims, sizeof(int) * nb_dims);
  desc.stride[nb_dims - 1] = 1;
  for (int i = nb_dims - 2; i >= 0; --i) {
    desc.stride[i] = desc.stride[i + 1] * desc.shape[i + 1];
  }
}

template <typename T>
void grid_sample(T *output, const T *input, const T *grid, int *output_dims, int *input_dims,
                 int *grid_dims,
                 GridSamplerPadding padding, bool align_corners, hipStream_t stream) {
  TensorDesc input_desc;
  create_desc(input_dims, 4, input_desc);

  TensorDesc output_desc;
  create_desc(output_dims, 4, output_desc);

  TensorDesc grid_desc;
  create_desc(grid_dims, 4, grid_desc);

  int count = 1;
  for (int i = 0; i < 4; ++i) {
    if (i == 1) {
      continue;
    }
    count *= output_desc.shape[i];
  }

  grid_sampler_2d_kernel<T><<<GET_BLOCKS(count), NUM_THREADS, 0, stream>>>(
      count, input, grid, output, input_desc, grid_desc, output_desc, padding,
      align_corners);

}

template void grid_sample<float>(float *output, const float *input, const float *grid,
                                 int *output_dims, int *input_dims, int *grid_dims,
                                  GridSamplerPadding padding,
                                 bool align_corners, hipStream_t stream);


__global__ void compute_sample_grid_kernel(float* __restrict__ grid, 
                                           const float* __restrict__ transform, 
                                           int bev_w, int bev_h){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < bev_w * bev_h){
        int i = idx / bev_h;
        int j = idx % bev_h;
        float x = transform[0 * 3 + 0] * j + transform[0 * 3 + 1] * i + transform[0 * 3 + 2]; // * 1.0
        float y = transform[1 * 3 + 0] * j + transform[1 * 3 + 1] * i + transform[1 * 3 + 2]; // * 1.0

        grid[i * bev_h * 2 + j * 2 + 0] = x / (bev_w - 1.0f) * 2.0f - 1.0f;
        grid[i * bev_h * 2 + j * 2 + 1] = y / (bev_h - 1.0f) * 2.0f - 1.0f;
    }
}

void compute_sample_grid_cuda(float* grid_dev, const float* transform, int bev_w, int bev_h, 
                                                                        hipStream_t stream){
    /*
    grid_dev : bev_w * bev_h * 2
    */

    dim3 grid(DIVUP(bev_w * bev_h, NUM_THREADS));
    dim3 block(NUM_THREADS);
    compute_sample_grid_kernel<<<grid, block, 0, stream>>>(grid_dev, transform, bev_w, bev_h);
}